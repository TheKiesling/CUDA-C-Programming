/*---------------------------------------------------------------------------
* UNIVERSIDAD DEL VALLE DE GUATEMALA
* FACULTAD DE INGENIERIA
*
* Nombres:
* Jose Pablo Kiesling Lange - 21581
* Pablo Andres Zamora Vasquez - 21780
* Laboratorio No.06
------------------------------------------------------------------------------*/

#include <stdio.h>
#include <hip/hip_runtime.h>

/************************************************************************
 * Kernel ejecutado por Device
 * Opera cada elemento de los vectores A y B, guardandolo en la posición
 * correspondiente en C
 * @param A 
 * @param B 
 * @param C 
 * @param numElements 
 */
__global__ void
vectorMultiply(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements){
        C[i] = A[i] * B[i];
    }
}
//***********************************************************************


/************************************************************************
 * Rutina ejecutada por Host
 */
int main(void)
{
    //-------------------- 0. Variables de control--------------------
    hipError_t err = hipSuccess;
    int numElements = 768;
    size_t size = numElements * sizeof(float);
    float result = 0.0;
  
    //-------------------- 1.1 Reservar memoria en Host para vectores A B y C --------------------
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    if (h_A == NULL || h_B == NULL || h_C == NULL){
        fprintf(stderr, "No se ha podido reservar memoria en Host para los vectores\n");
        exit(EXIT_FAILURE);
    }

    //-------------------- 1.2 Reservar memoria en Device para vectores A B y C --------------------
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess){
        fprintf(stderr, "No se ha podido reservar memoria en Device para el vector A (codigo de error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess){
        fprintf(stderr, "No se ha podido reservar memoria en Device para el vector B (codigo de error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess){
        fprintf(stderr, "No se ha podido reservar memoria en Device para el vector C (codigo de error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    //-------------------- 2. Generar valores para operandos A B en Host --------------------
    for (int i = 0; i < numElements; ++i){
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    //-------------------- 3. Pasar valores de A & B de Host a Device --------------------
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess){
        fprintf(stderr, "No se ha podido copiar el vector A de Host a Device (codigo de error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess){
        fprintf(stderr, "No se ha podido copiar el vector B de Host a Device (codigo de error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //-------------------- 4. Ejecutar Kernel en DEVICE --------------------
    int threadsPerBlock = 768;
    int blocks = 1;
    vectorMultiply<<< blocks, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess){
        fprintf(stderr, "No se ha podido ejecutar el Kernel vectorMultiply (codigo de error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
 
    //-------------------- 5.1 Copiar resultado de DEVICE a HOST --------------------
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess){
        fprintf(stderr, "No se ha podido copiar el vector C de Device a Host (codigo de error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //-------------------- 5.2 Sumar secuencialmente los elementos del vector C --------------------
    for (int i = 0; i < numElements; i++){
        result += h_C[i];
        printf("Producto de los elementos [%d] de A & B: %f\n", i, h_C[i]);
    }
    printf("\n----------------------------------------\n");
    printf("RESULTADO FINAL del producto punto: %f", result);

    
    //-------------------- 6. Liberar memoria del HOST --------------------
    free(h_A);
    free(h_B);
    free(h_C);
 
    //-------------------- 7. Liberar memoria Global --------------------
    err = hipFree(d_A);
    if (err != hipSuccess){
        fprintf(stderr, "No se ha podido liberar la memoria del vector A (codigo de error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);
    if (err != hipSuccess){
        fprintf(stderr, "No se ha podido liberar la memoria del vector B (codigo de error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipFree(d_C);
    if (err != hipSuccess){
        fprintf(stderr, "No se ha podido liberar la memoria del vector C (codigo de error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipDeviceReset();
    if (err != hipSuccess){
        fprintf(stderr, "No se ha podido reiniciar el Device (codigo de error: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
 
    return 0;
}

